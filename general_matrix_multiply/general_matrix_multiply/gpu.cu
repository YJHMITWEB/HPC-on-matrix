#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <iostream>
#include <string>
using namespace std;

#include "gpu.h"
#include "config.h"

__global__ void gpu_common_kernel(int M_, int K_, int N_, const float4* __restrict__ A, const float* __restrict__ B, float4* __restrict__ result)
{

}

float* gpu_common(const float* m1, const float* m2, config &c)
{
	hipEvent_t start, stop;
	hipEventCreate(&start, 0);
	hipEventCreate(&stop, 0);
	hipEventRecord(start, 0);

	float* result = (float*)malloc(sizeof(float) * c.M1_HEIGHT * c.M2_WIDTH);
	//gpu_common_kernel << < 4, 4 >> > (M, K, N, m1, m2, result);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("gpu common --- Time to compute:  %3.8f ms\n", elapsedTime);

	return result;
}

void run_gpu(float * m1, float * m2, config & c)
{
	float* gpu_common_result = gpu_common(m1, m2, c);
}
